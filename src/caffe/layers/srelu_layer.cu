#include "hip/hip_runtime.h"
#include <algorithm>
#include <vector>

#include "caffe/layers/neuron_layer.hpp"
#include "caffe/layers/srelu_layer.hpp"

namespace caffe {

// CUDA kernele for forward
template <typename Dtype>
__global__ void SReLUForward(const int n, const int channels, const int dim,
    const Dtype* in, Dtype* out,
    const Dtype* tr_data, const Dtype* ar_data, const Dtype* tl_data, const Dtype* al_data,
    const int div_factor) {
  CUDA_KERNEL_LOOP(index, n) {
    int c = (index / dim) % channels / div_factor;
    if (in[index] >= tr_data[c])
    	out[index] = tr_data[c] + ar_data[c] * (in[index] - tr_data[c]);
    else if ( in[index] > tl_data[c] && in[index] < tr_data[c] )
    	out[index] = in[index];
    else
    	out[index] = tl_data[c] + al_data[c] * (in[index] - tl_data[c]);    
  }
}


// CUDA kernel for bottom backward
template <typename Dtype>
__global__ void SReLUBackward(const int n, const int channels, const int dim,
    const Dtype* in_diff, const Dtype* in_data, Dtype* out_diff,
    const Dtype* tr_data, const Dtype* ar_data, const Dtype* tl_data, const Dtype* al_data,
    const int div_factor) {
  CUDA_KERNEL_LOOP(index, n) {
    int c = (index / dim) % channels / div_factor;
    out_diff[index] = in_diff[index] * 
    	(ar_data[c] * (in_data[index] >= tr_data[c]) + 
    	 (in_data[index] > tl_data[c] && in_data[index] < tr_data[c]) +
         al_data[c] * (in_data[index] <= tl_data[c]));
  }
}


// CUDA kernel for element-wise parameter backward
template <typename Dtype>
__global__ void SReLUParamsBackward(const int n,
    const int rows, const int dim, const int rowPitch,
    const Dtype* in_diff, const Dtype* in_data, 
    const Dtype* tr_data, const Dtype* ar_data, const Dtype* tl_data, const Dtype* al_data,
    Dtype* out_diff,
    int param_num) {
  switch (param_num) {
    case 0:
      CUDA_KERNEL_LOOP(index, n) {
        out_diff[index] = in_diff[index] * (1 - ar_data[index/dim]) * (in_data[index] >= tr_data[index/dim]);
        for ( int k = 1; k < rows; k++ ) {
            out_diff[index] += in_diff[index + k*rowPitch]
               * (1 - ar_data[index/dim]) * (in_data[index + k*rowPitch] >= tr_data[index/dim]);
        }
      }
      break;
    case 1:
      CUDA_KERNEL_LOOP(index, n) {
        out_diff[index] = in_diff[index] * (in_data[index] - tr_data[index/dim]) * (in_data[index] >= tr_data[index/dim]);
        for ( int k = 1; k < rows; k++ ) {
            out_diff[index] += in_diff[index + k*rowPitch]
               * (in_data[index + k*rowPitch] - tr_data[index/dim]) * (in_data[index + k*rowPitch] >= tr_data[index/dim]);
        }
      }
      break;
    case 2:
      CUDA_KERNEL_LOOP(index, n) {
        out_diff[index] = in_diff[index] * (1 - al_data[index/dim]) * (in_data[index] <= tl_data[index/dim]);
        for ( int k = 1; k < rows; k++ ) {
            out_diff[index] += in_diff[index + k*rowPitch]
               * (1 - al_data[index/dim]) * (in_data[index + k*rowPitch] <= tl_data[index/dim]);
        }
      }
      break;
    case 3:            
      CUDA_KERNEL_LOOP(index, n) {        
        out_diff[index] = in_diff[index] * (in_data[index] - tl_data[index/dim]) * (in_data[index] <= tl_data[index/dim]);
        for ( int k = 1; k < rows; k++ ) {
            out_diff[index] += in_diff[index + k*rowPitch]
               * (in_data[index + k*rowPitch] - tl_data[index/dim]) * (in_data[index + k*rowPitch] <= tl_data[index/dim]);                
        }
      }
      break;
    default:
      assert(false);
  }
}

template <typename Dtype>
void SReLULayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  const int count = bottom[0]->count();
  const int dim = bottom[0]->count(2);
  const int channels = bottom[0]->channels();
  const Dtype* tr_data = this->blobs_[0]->gpu_data();
  const Dtype* ar_data = this->blobs_[1]->gpu_data();
  const Dtype* tl_data = this->blobs_[2]->gpu_data();
  const Dtype* al_data = this->blobs_[3]->gpu_data();
  const int div_factor = channel_shared_ ? channels : 1;

  // For in-place computation
  if (top[0] == bottom[0]) {
    caffe_copy(count, bottom_data, bottom_memory_.mutable_gpu_data());
  }

  // NOLINT_NEXT_LINE(whitespace/operators)
  SReLUForward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
      count, channels, dim, bottom_data, top_data,
      tr_data, ar_data, tl_data, al_data, div_factor);
  CUDA_POST_KERNEL_CHECK;
}


template <typename Dtype>
void SReLULayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down,
    const vector<Blob<Dtype>*>& bottom) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  const Dtype* top_diff = top[0]->gpu_diff();
  const Dtype* tr_data = this->blobs_[0]->gpu_data();
  const Dtype* ar_data = this->blobs_[1]->gpu_data();
  const Dtype* tl_data = this->blobs_[2]->gpu_data();
  const Dtype* al_data = this->blobs_[3]->gpu_data();
  const int count = bottom[0]->count();
  const int cdim = bottom[0]->count(1);
  const int dim = bottom[0]->count(2);      
  const int channels = bottom[0]->channels();

  // For in-place computation
  if (top[0] == bottom[0]) {
    bottom_data = bottom_memory_.gpu_data();
  }

  // Propagate to param
  // Since to write bottom diff will affect top diff if top and bottom blobs
  // are identical (in-place computaion), we first compute param backward to
  // keep top_diff unchanged.    
  for (int param_num = 0; param_num < 4; ++param_num) {    
    if (this->param_propagate_down_[param_num]) {
      Dtype* param_diff = this->blobs_[param_num]->mutable_gpu_diff();          
      // compute element-wise diff
      // NOLINT_NEXT_LINE(whitespace/operators)
      SReLUParamsBackward<Dtype><<<CAFFE_GET_BLOCKS(cdim),
        CAFFE_CUDA_NUM_THREADS>>>(
        cdim, bottom[0]->num(), dim, top[0]->offset(1),
        top_diff, bottom_data,
        tr_data, ar_data, tl_data, al_data,
        backward_buff_.mutable_gpu_diff(),
        param_num);      
      CUDA_POST_KERNEL_CHECK;
      if (channel_shared_) {
        Dtype dsum;
        caffe_gpu_dot<Dtype>(channels * dim, backward_buff_.gpu_diff(),
         multiplier_.gpu_data(), &dsum);
        caffe_gpu_add_scalar(this->blobs_[0]->count(), Dtype(dsum), param_diff);
      } else {
        caffe_gpu_gemv<Dtype>(CblasNoTrans, channels, dim, 1.,
          backward_buff_.gpu_diff(), multiplier_.gpu_data(), 1.,
          param_diff);
      }
    }
  }
  
  // Propagate to bottom
  if (propagate_down[0]) {
    Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
    const Dtype* slope_data = this->blobs_[0]->gpu_data();
    int div_factor = channel_shared_ ? channels : 1;
    // NOLINT_NEXT_LINE(whitespace/operators)
    SReLUBackward<Dtype><<<CAFFE_GET_BLOCKS(count),
        CAFFE_CUDA_NUM_THREADS>>>(
        count, channels, dim, top_diff, bottom_data, bottom_diff,
        tr_data, ar_data, tl_data, al_data,
        div_factor);
    CUDA_POST_KERNEL_CHECK;
  }
}


INSTANTIATE_LAYER_GPU_FUNCS(SReLULayer);

}  // namespace caffe
